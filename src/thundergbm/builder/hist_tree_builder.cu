#include "hip/hip_runtime.h"
//
// Created by ss on 19-1-20.
//
#include "thundergbm/builder/hist_tree_builder.h"

#include "thundergbm/util/cub_wrapper.h"
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/iterator/counting_iterator.h"
#include "thrust/iterator/transform_iterator.h"
#include "thrust/iterator/discard_iterator.h"
#include "thrust/sequence.h"
#include "thrust/binary_search.h"
#include "thundergbm/util/multi_device.h"

void HistTreeBuilder::get_bin_ids() {
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        SparseColumns &columns = shards[device_id].columns;
        HistCut &cut = this->cut[device_id];
        auto &dense_bin_id = this->dense_bin_id[device_id];
        using namespace thrust;
        int n_column = columns.n_column;
        int nnz = columns.nnz;
        auto cut_row_ptr = cut.cut_row_ptr.device_data();
        auto cut_points_ptr = cut.cut_points_val.device_data();
        auto csc_val_data = columns.csc_val.device_data();
        SyncArray<unsigned char> bin_id;
        bin_id.resize(columns.nnz);
        auto bin_id_data = bin_id.device_data();
        int n_block = fminf((nnz / n_column - 1) / 256 + 1, 4 * 56);
        {
            auto lowerBound = [=]__device__(const float_type *search_begin, const float_type *search_end, float_type val) {
                const float_type *left = search_begin;
                const float_type *right = search_end;

                while (left != right) {
                    const float_type *mid = left + (right - left) / 2;
                    if (*mid <= val)
                        right = mid;
                    else left = mid + 1;
                }
                return left;
            };
            TIMED_SCOPE(timerObj, "binning");
            device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int cid, int i) {
                auto search_begin = cut_points_ptr + cut_row_ptr[cid];
                auto search_end = cut_points_ptr + cut_row_ptr[cid + 1];
                auto val = csc_val_data[i];
                bin_id_data[i] = lowerBound(search_begin, search_end, val) - search_begin;
            }, n_block);
        }

        auto max_num_bin = param.max_num_bin;
        dense_bin_id.resize(n_instances * n_column);
        auto dense_bin_id_data = dense_bin_id.device_data();
        auto csc_row_idx_data = columns.csc_row_idx.device_data();
        device_loop(n_instances * n_column, [=]__device__(int i) {
        dense_bin_id_data[i] = max_num_bin;
    });
        device_loop_2d(n_column, columns.csc_col_ptr.device_data(), [=]__device__(int fid, int i) {
        int row = csc_row_idx_data[i];
        unsigned char bid = bin_id_data[i];
        dense_bin_id_data[row * n_column + fid] = bid;
    }, n_block);
    });
}

void HistTreeBuilder::find_split(int level, int device_id) {
    const SparseColumns &columns = shards[device_id].columns;
    SyncArray<int> &nid = ins2node_id[device_id];
    SyncArray<GHPair> &gh_pair = gradients[device_id];
    Tree &tree = trees[device_id];
    SyncArray<SplitPoint> &sp = this->sp[device_id];
    SyncArray<bool> &ignored_set = shards[device_id].ignored_set;
    HistCut &cut = this->cut[device_id];
    auto &dense_bin_id = this->dense_bin_id[device_id];
    auto &last_hist = this->last_hist[device_id];

    TIMED_FUNC(timerObj);
    int n_nodes_in_level = static_cast<int>(pow(2, level));
    int nid_offset = static_cast<int>(pow(2, level) - 1);
    int n_column = columns.n_column;
    int n_partition = n_column * n_nodes_in_level;
    int n_bins = cut.cut_points.size();
    int n_max_nodes = 2 << param.depth;
    int n_max_splits = n_max_nodes * n_bins;
    int n_split = n_nodes_in_level * n_bins;

    LOG(TRACE) << "start finding split";

    //find the best split locally
    {
        using namespace thrust;

        //calculate split information for each split
        SyncArray<GHPair> hist(n_max_splits);
        SyncArray<GHPair> missing_gh(n_partition);
        auto cut_fid_data = cut.cut_fid.device_data();
        auto i2fid = [=] __device__(int i) { return cut_fid_data[i % n_bins]; };
        auto hist_fid = make_transform_iterator(counting_iterator<int>(0), i2fid);
        {
            {
                TIMED_SCOPE(timerObj, "build hist");
                {
                    size_t
                    smem_size = n_bins * sizeof(GHPair);
                    LOG(DEBUG) << "shared memory size = " << smem_size / 1024.0 << " KB";
                    if (n_nodes_in_level == 1) {
                        //root
                        auto hist_data = hist.device_data();
                        auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                        auto gh_data = gh_pair.device_data();
                        auto dense_bin_id_data = dense_bin_id.device_data();
                        auto max_num_bin = param.max_num_bin;
                        auto n_instances = this->n_instances;
                        if (smem_size > 48 * 1024) {
                            device_loop(n_instances * n_column, [=]__device__(int i) {
                                int iid = i / n_column;
                                int fid = i % n_column;
                                unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                if (bid != max_num_bin) {
                                    int feature_offset = cut_row_ptr_data[fid];
                                    const GHPair src = gh_data[iid];
                                    GHPair &dest = hist_data[feature_offset + bid];
                                    atomicAdd(&dest.g, src.g);
                                    atomicAdd(&dest.h, src.h);
                                }
                            });
                        } else {
                            anonymous_kernel([=]__device__() {
                                extern __shared__ GHPair local_hist[];
                                for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                    local_hist[i] = 0;
                                }
                                __syncthreads();
                                for (int i = blockIdx.x * blockDim.x + threadIdx.x;
                                     i < n_instances * n_column; i += blockDim.x * gridDim.x) {
                                    int iid = i / n_column;
                                    int fid = i % n_column;
                                    unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                    if (bid != max_num_bin) {
                                        int feature_offset = cut_row_ptr_data[fid];
                                        const GHPair src = gh_data[iid];
                                        GHPair &dest = local_hist[feature_offset + bid];
                                        atomicAdd(&dest.g, src.g);
                                        atomicAdd(&dest.h, src.h);
                                    }
                                }
                                __syncthreads();
                                for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                    GHPair &dest = hist_data[i];
                                    GHPair src = local_hist[i];
                                    atomicAdd(&dest.g, src.g);
                                    atomicAdd(&dest.h, src.h);
                                }
                            }, smem_size);
                        }
                    } else {
                        //otherwise
                        SyncArray<int> node_idx(n_instances);
                        SyncArray<int> node_ptr(n_nodes_in_level + 1);
                        {
                            TIMED_SCOPE(timerObj, "data partitioning");
                            SyncArray<int> nid4sort(n_instances);
                            nid4sort.copy_from(ins2node_id[device_id]);
                            sequence(cuda::par, node_idx.device_data(), node_idx.device_end(), 0);
                            cub_sort_by_key(nid4sort, node_idx);
                            auto counting_iter = make_counting_iterator < int > (nid_offset);
                            node_ptr.host_data()[0] =
                                    lower_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), nid_offset) -
                                    nid4sort.device_data();

                            upper_bound(cuda::par, nid4sort.device_data(), nid4sort.device_end(), counting_iter,
                                        counting_iter + n_nodes_in_level, node_ptr.device_data() + 1);
                            LOG(DEBUG) << "node ptr = " << node_ptr;
                            hipDeviceSynchronize();
                        }

                        for (int i = 0; i < n_nodes_in_level / 2; ++i) {
                            int nid0_to_compute = i * 2;
                            int nid0_to_substract = i * 2 + 1;
                            auto node_ptr_data = node_ptr.host_data();
                            int n_ins_left = node_ptr_data[nid0_to_compute + 1] - node_ptr_data[nid0_to_compute];
                            int n_ins_right = node_ptr_data[nid0_to_substract + 1] - node_ptr_data[nid0_to_substract];
                            if (max(n_ins_left, n_ins_right) == 0) continue;
                            if (n_ins_left > n_ins_right)
                                swap(nid0_to_compute, nid0_to_substract);

                            //compute
                            {
                                int nid0 = nid0_to_compute;
                                auto idx_begin = node_ptr.host_data()[nid0];
                                auto idx_end = node_ptr.host_data()[nid0 + 1];
                                auto node_idx_data = node_idx.device_data();
                                auto hist_data = hist.device_data() + nid0 * n_bins;
                                auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
                                auto gh_data = gh_pair.device_data();
                                auto dense_bin_id_data = dense_bin_id.device_data();
                                auto max_num_bin = param.max_num_bin;
                                if (smem_size > 48 * 1024) {
                                    device_loop((idx_end - idx_begin) * n_column, [=]__device__(int i) {
                                        int iid = node_idx_data[i / n_column + idx_begin];
                                        int fid = i % n_column;
                                        unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                        if (bid != max_num_bin) {
                                            int feature_offset = cut_row_ptr_data[fid];
                                            const GHPair src = gh_data[iid];
                                            GHPair &dest = hist_data[feature_offset + bid];
                                            atomicAdd(&dest.g, src.g);
                                            atomicAdd(&dest.h, src.h);
                                        }
                                    });
                                } else {
                                    anonymous_kernel([=]__device__() {
                                        extern __shared__ GHPair local_hist[];
                                        for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                            local_hist[i] = 0;
                                        }
                                        __syncthreads();

                                        for (int i = blockIdx.x * blockDim.x + threadIdx.x;
                                             i < (idx_end - idx_begin) * n_column; i += blockDim.x * gridDim.x) {
                                            int iid = node_idx_data[i / n_column + idx_begin];
                                            int fid = i % n_column;
                                            unsigned char bid = dense_bin_id_data[iid * n_column + fid];
                                            if (bid != max_num_bin) {
                                                int feature_offset = cut_row_ptr_data[fid];
                                                const GHPair src = gh_data[iid];
                                                GHPair &dest = local_hist[feature_offset + bid];
                                                atomicAdd(&dest.g, src.g);
                                                atomicAdd(&dest.h, src.h);
                                            }
                                        }
                                        __syncthreads();
                                        for (int i = threadIdx.x; i < n_bins; i += blockDim.x) {
                                            GHPair &dest = hist_data[i];
                                            GHPair src = local_hist[i];
                                            atomicAdd(&dest.g, src.g);
                                            atomicAdd(&dest.h, src.h);
                                        }
                                    }, smem_size);
                                }
                            }

                            //substract
                            {
                                auto hist_data_computed = hist.device_data() + nid0_to_compute * n_bins;
                                auto hist_data_to_compute = hist.device_data() + nid0_to_substract * n_bins;
                                auto father_hist_data = last_hist.device_data() + (nid0_to_substract / 2) * n_bins;
                                device_loop(n_bins, [=]__device__(int i) {
                                    hist_data_to_compute[i] = father_hist_data[i] - hist_data_computed[i];
                                });
                            }
//                            PERFORMANCE_CHECKPOINT(timerObj);
                        }
                    }
                    last_hist.copy_from(hist);
                    hipDeviceSynchronize();
                }
                LOG(DEBUG) << "hist new = " << hist;
                LOG(DEBUG) << "cutfid = " << cut.cut_fid;
                inclusive_scan_by_key(cuda::par, hist_fid, hist_fid + n_split,
                                      hist.device_data(), hist.device_data());
                LOG(DEBUG) << hist;

                auto nodes_data = tree.nodes.device_data();
                auto missing_gh_data = missing_gh.device_data();
                auto cut_row_ptr = cut.cut_row_ptr.device_data();
                auto hist_data = hist.device_data();
                device_loop(n_partition, [=]__device__(int pid) {
                    int nid0 = pid / n_column;
                    int nid = nid0 + nid_offset;
                    if (!nodes_data[nid].splittable()) return;
                    int fid = pid % n_column;
                    if (cut_row_ptr[fid + 1] != cut_row_ptr[fid]) {
                        GHPair node_gh = hist_data[nid0 * n_bins + cut_row_ptr[fid + 1] - 1];
                        missing_gh_data[pid] = nodes_data[nid].sum_gh_pair - node_gh;
                    }
                });
                LOG(DEBUG) << missing_gh;
            }
        }
        //calculate gain of each split
        SyncArray<float_type> gain(n_max_splits);
        {
//            TIMED_SCOPE(timerObj, "calculate gain");
            auto compute_gain = []__device__(GHPair father, GHPair lch, GHPair rch, float_type min_child_weight,
                    float_type lambda) -> float_type {
                    if (lch.h >= min_child_weight && rch.h >= min_child_weight)
                    return (lch.g * lch.g) / (lch.h + lambda) + (rch.g * rch.g) / (rch.h + lambda) -
            (father.g * father.g) / (father.h + lambda);
                    else
                    return 0;
            };

            const Tree::TreeNode *nodes_data = tree.nodes.device_data();
            GHPair *gh_prefix_sum_data = hist.device_data();
            float_type *gain_data = gain.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            auto ignored_set_data = ignored_set.device_data();
            //for lambda expression
            float_type mcw = param.min_child_weight;
            float_type l = param.lambda;
            device_loop(n_split, [=]__device__(int i) {
                int nid0 = i / n_bins;
                int nid = nid0 + nid_offset;
                int fid = hist_fid[i % n_bins];
                if (nodes_data[nid].is_valid && !ignored_set_data[fid]) {
                    int pid = nid0 * n_column + hist_fid[i];
                    GHPair father_gh = nodes_data[nid].sum_gh_pair;
                    GHPair p_missing_gh = missing_gh_data[pid];
                    GHPair rch_gh = gh_prefix_sum_data[i];
                    float_type default_to_left_gain = max(0.f,
                                                          compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                    rch_gh = rch_gh + p_missing_gh;
                    float_type default_to_right_gain = max(0.f,
                                                           compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                    if (default_to_left_gain > default_to_right_gain)
                        gain_data[i] = default_to_left_gain;
                    else
                        gain_data[i] = -default_to_right_gain;//negative means default split to right

                } else gain_data[i] = 0;
            });
            LOG(DEBUG) << "gain = " << gain;
        }

        SyncArray<int_float> best_idx_gain(n_nodes_in_level);
        {
//            TIMED_SCOPE(timerObj, "get best gain");
            auto arg_abs_max = []__device__(const int_float &a, const int_float &b) {
                if (fabsf(get<1>(a)) == fabsf(get<1>(b)))
                    return get<0>(a) < get<0>(b) ? a : b;
                else
                    return fabsf(get<1>(a)) > fabsf(get<1>(b)) ? a : b;
            };

            auto nid_iterator = make_transform_iterator(counting_iterator<int>(0), placeholders::_1 / n_bins);

            reduce_by_key(
                    cuda::par,
                    nid_iterator, nid_iterator + n_split,
                    make_zip_iterator(make_tuple(counting_iterator<int>(0), gain.device_data())),
                    make_discard_iterator(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max
            );
            LOG(DEBUG) << n_split;
            LOG(DEBUG) << "best rank & gain = " << best_idx_gain;
        }

        //get split points
        {
            const int_float *best_idx_gain_data = best_idx_gain.device_data();
            auto hist_data = hist.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            auto cut_val_data = cut.cut_points_val.device_data();

            sp.resize(n_nodes_in_level);
            auto sp_data = sp.device_data();
            auto nodes_data = tree.nodes.device_data();

            int column_offset = columns.column_offset;

            auto cut_row_ptr_data = cut.cut_row_ptr.device_data();
            device_loop(n_nodes_in_level, [=]__device__(int i) {
                int_float bst = best_idx_gain_data[i];
                float_type best_split_gain = get<1>(bst);
                int split_index = get<0>(bst);
                if (!nodes_data[i + nid_offset].is_valid) {
                    sp_data[i].split_fea_id = -1;
                    sp_data[i].nid = -1;
                    return;
                }
                int fid = hist_fid[split_index];
                sp_data[i].split_fea_id = fid + column_offset;
                sp_data[i].nid = i + nid_offset;
                sp_data[i].gain = fabsf(best_split_gain);
                sp_data[i].fval = cut_val_data[split_index % n_bins];
                sp_data[i].split_bid = (unsigned char) (split_index % n_bins - cut_row_ptr_data[fid]);
                sp_data[i].fea_missing_gh = missing_gh_data[i * n_column + hist_fid[split_index]];
                sp_data[i].default_right = best_split_gain < 0;
                sp_data[i].rch_sum_gh = hist_data[split_index];
            });
        }
    }

    LOG(DEBUG) << "split points (gain/fea_id/nid): " << sp;
}

void HistTreeBuilder::update_ins2node_id() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        SyncArray<bool> has_splittable(1);
        auto &columns = shards[device_id].columns;
        //set new node id for each instance
        {
//        TIMED_SCOPE(timerObj, "get new node id");
            auto nid_data = ins2node_id[device_id].device_data();
            const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
            has_splittable.host_data()[0] = false;
            bool *h_s_data = has_splittable.device_data();
            int column_offset = columns.column_offset;

            int n_column = columns.n_column;
            auto dense_bin_id_data = dense_bin_id[device_id].device_data();
            int max_num_bin = param.max_num_bin;
            device_loop(n_instances, [=]__device__(int iid) {
                int nid = nid_data[iid];
                const Tree::TreeNode &node = nodes_data[nid];
                int split_fid = node.split_feature_id;
                if (node.splittable() && ((split_fid - column_offset < n_column) && (split_fid >= column_offset))) {
                    h_s_data[0] = true;
                    unsigned char split_bid = node.split_bid;
                    unsigned char bid = dense_bin_id_data[iid * n_column + split_fid - column_offset];
                    bool to_left = true;
                    if ((bid == max_num_bin && node.default_right) || (bid <= split_bid))
                        to_left = false;
                    if (to_left) {
                        //goes to left child
                        nid_data[iid] = node.lch_index;
                    } else {
                        //right child
                        nid_data[iid] = node.rch_index;
                    }
                }
            });
        }
        LOG(DEBUG) << "new tree_id = " << ins2node_id[device_id];
        has_split[device_id] = has_splittable.host_data()[0];
    });
}

void HistTreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    TreeBuilder::init(dataset, param);
    //TODO refactor
    //init shards
    int n_device = param.n_device;
    shards = vector<Shard>(n_device);
    vector<std::unique_ptr<SparseColumns>> v_columns(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].reset(&shards[i].columns);
        shards[i].ignored_set = SyncArray<bool>(n_instances);
    }
    SparseColumns columns;
    columns.from_dataset(dataset);
    columns.to_multi_devices(v_columns);
    cut = vector<HistCut>(param.n_device);
    dense_bin_id = MSyncArray<unsigned char>(param.n_device);
    last_hist = MSyncArray<GHPair>(param.n_device);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        cut[device_id].get_cut_points2(shards[device_id].columns, param.max_num_bin, n_instances);
        last_hist[device_id].resize((2 << param.depth) * cut[device_id].cut_points.size());
    });
    get_bin_ids();
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].release();
    }
    SyncMem::clear_cache();
}
