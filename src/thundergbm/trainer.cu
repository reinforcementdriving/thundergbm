#include "hip/hip_runtime.h"
//
// Created by zeyi on 1/9/19.
//
#include <fstream>
#include "hip/hip_runtime_api.h"
#include <boost/archive/text_oarchive.hpp>
#include <boost/serialization/vector.hpp>

#include <thundergbm/tree.h>
#include <thundergbm/trainer.h>
#include <thundergbm/metric/metric.h>
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/reduce.h"
#include "time.h"
#include "thundergbm/booster.h"
#include "chrono"

vector<vector<Tree>> TreeTrainer::train(GBMParam &param, const DataSet &dataset) {
    if (param.tree_method == "auto")
        if (dataset.n_features() > 20000)
            param.tree_method = "exact";
        else
            param.tree_method = "hist";

    //correct the number of classes
    if(param.objective.find("multi:") != std::string::npos) {
        int num_class = dataset.label.size();
        if (param.num_class != num_class) {
            LOG(INFO) << "updating number of classes from " << param.num_class << " to " << num_class;
            param.num_class = num_class;
        }
    }
    else if(param.objective.find("reg:") != std::string::npos){
        param.num_class = 1;
    }

    vector<vector<Tree>> boosted_model;
    Booster booster;
    booster.init(dataset, param);
    std::chrono::high_resolution_clock timer;
    auto start = timer.now();
    for (int i = 0; i < param.n_trees; ++i) {
        //one iteration may produce multiple trees, depending on objectives
        booster.boost(boosted_model);
    }
    auto stop = timer.now();
    std::chrono::duration<float> training_time = stop - start;
    LOG(INFO) << "training time = " << training_time.count();

    //save model
    std::ofstream ofs;
	ofs.open(param.out_model_name, std::ofstream::trunc);
    boost::archive::text_oarchive oa(ofs);
    oa & param.objective;
    oa & param.learning_rate;
    oa & param.num_class;
    oa & dataset.label;
    //oa & param;
    oa & boosted_model;
    ofs.close();
	SyncMem::clear_cache();
	return boosted_model;
}
